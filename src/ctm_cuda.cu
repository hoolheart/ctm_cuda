#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "ctm_cuda.h"

#define BLOCK_SIZE 256

CtmCell *dCells;
CtmLink *dLinks;
float *dPosIn;
float *dPosOut;
float *dIn;
float *dOut;
int numCells;

//__device__ float min(float d1, float d2) {
//	if (d1<d2)
//		return d1;
//	else
//		return d2;
//}

__device__ float mid(float d1, float d2, float d3) {
	if (d1<=d2) {
		if (d2<=d3)
			return d2;
		else {
			if (d1<=d3)
				return d3;
			else
				return d1;
		}
	}
	else {
		if (d1<=d3)
			return d1;
		else {
			if (d2<=d3)
				return d3;
			else
				return d2;
		}
	}
}

__global__ void updateAccess(CtmCell *c, int *a, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {}

    __syncthreads();
}

__global__ void calPosFlow(CtmCell *ListCell, float *CellPosIn, float *CellPosOut, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i<n) {
		switch (ListCell[i].type) {
		case CELL_TYPE_INPUT:
			CellPosIn[i] = 0;
			ListCell[i].length += ListCell[i].rate*dt;
			CellPosOut[i] = ListCell[i].length;
			break;
		case CELL_TYPE_OUTPUT:
			CellPosIn[i] = ListCell[i].rate*dt;
			CellPosOut[i] = 0;
			break;
		case CELL_TYPE_NORMAL:
			break;
		default:
			break;
		}
    }

    __syncthreads();
}

__global__ void calFlow(
		CtmCell *ListCell,
		CtmLink *ListLink,
		float *CellPosIn,
		float *CellPosOut,
		float *CellIn,
		float *CellOut,
		int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<n)
		CellOut[i] = 0;
	if (i<n) {
	}

    __syncthreads();
}

__global__ void updateCells(
		CtmCell *ListCell,
		float *CellIn,
		float *CellOut,
		int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<n)
		ListCell[i].length += CellIn[i]-CellOut[i];

    __syncthreads();
}

void deleteCudaEnv() {
	hipFree(dCells);
	hipFree(dLinks);
	hipFree(dPosIn);
	hipFree(dPosOut);
	hipFree(dIn);
	hipFree(dOut);
}

void createCudaEnv(
		CtmCell *hCells,
		CtmLink *hLinks,
		float *hPosIn,
		float *hPosOut,
		float *hIn,
		float *hOut,
		int n) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Create CUDA variables
    numCells = n;
    err = hipMalloc((void **)&dCells, n*sizeof(CtmCell));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&dLinks, n*sizeof(CtmLink));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&dPosIn, n*sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&dPosOut, n*sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&dIn, n*sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&dOut, n*sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy host data to device
    err = hipMemcpy(dCells, hCells, n*sizeof(CtmCell), hipMemcpyHostToDevice);
    if (err != hipSuccess)     {
        fprintf(stderr, "Failed to copy host data to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dLinks, hLinks, n*sizeof(CtmLink), hipMemcpyHostToDevice);
    if (err != hipSuccess)     {
        fprintf(stderr, "Failed to copy host data to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dPosIn, hPosIn, n*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)     {
        fprintf(stderr, "Failed to copy host data to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dPosOut, hPosOut, n*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)     {
        fprintf(stderr, "Failed to copy host data to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dIn, hIn, n*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)     {
        fprintf(stderr, "Failed to copy host data to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dOut, hOut, n*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)     {
        fprintf(stderr, "Failed to copy host data to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void updateCudaAcc(CtmCell *hCells) {
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // size of vector of accesses
    size_t size = numCells * sizeof(int);

    // Allocate the host input vector A
    int *h_A = (int *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host access data
    for (int i = 0; i < numCells; ++i) {
    }

    // Allocate the device access data
    int *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host access data to the device
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //update access data
    int blocksPerGrid=(numCells + BLOCK_SIZE - 1) / BLOCK_SIZE;
    updateAccess<<<blocksPerGrid, BLOCK_SIZE>>>(dCells,d_A,numCells);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // free all temporary data
    hipFree(d_A);
    free(h_A);
}

void loadCudaLen(CtmCell *hCells) {
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Copy the device cell data to the host
    err = hipMemcpy(hCells, dCells, numCells*sizeof(CtmCell), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy cell data from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

bool simCuda(float dt) {
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // calculate possible maximum flows
    int blocksPerGrid=(numCells + BLOCK_SIZE - 1) / BLOCK_SIZE;
    calPosFlow<<<blocksPerGrid, BLOCK_SIZE>>>(dCells,dPosIn,dPosOut,dt,numCells);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to calculate possible maximum flows (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // calculate actual flows
    calFlow<<<blocksPerGrid, BLOCK_SIZE>>>(dCells,dLinks,dPosIn,dPosOut,dIn,dOut,numCells);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to calculate actual flows (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // update cell lengths
    updateCells<<<blocksPerGrid, BLOCK_SIZE>>>(dCells,dIn,dOut,numCells);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to update the cell lengths (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	return true;
}
